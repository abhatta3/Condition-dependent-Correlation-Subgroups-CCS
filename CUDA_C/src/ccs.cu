#include "hip/hip_runtime.h"
/**************************************************************************
 * Condition-dependent Correlation Subgroups (CCS) 
 * Compilation: nvcc ccs.cu -o ccs_cuda -lm
 * Description: Biclustering has been emerged as a powerful tool for 
                identification of a group of co-expressed genes under a subset 
                of experimental conditions (measurements) present in a gene 
                expression dataset.  In this program we implemented CCS biclustering. 
 * Developer: Dr. Anindya Bhattacharya and Dr. Yan Cui, UTHSC, Memphis, TN, USA
 * Email: anindyamail123@gmail.com; ycui2@uthsc.edu 

Note: The minimum number of genes and the samples per bicluster is 10. 
User can alter the minimum size by changing the values for 'mingene' 
and 'min' defined in "ccs.h" file for minimum number of genes and samples
respectively. 

****************************************************************************
*/


#include "ccs.h"
#include "matrixsize.c"
#include "readgene.c"
#include "pair_cor.c"
#include "bicluster_pair_score.c"
#include "merge_bicluster.c"
#include "print_bicluster.c"
#define MAXSAMPLE 200 //number of samples in the input datamatrix. Fixed here to make static shared memory in global and device function. They dynamic alternative is commented out.


__device__  struct pair_r comput_r_cuda(char *sample,int wid,int k,int i,int D,float **gene)
{

	float sx, sxx, sy, sxy, syy;
	float sx_n, sxx_n, sy_n, sxy_n, syy_n;
	int j;
        __shared__ float genekj[MAXSAMPLE];
       __shared__ float geneij[MAXSAMPLE];
  
        struct pair_r rval;

        rval.r=0.0;
        rval.n_r=0.0; 

	sx = 0; sxx = 0; sy = 0; sxy = 0; syy = 0;

	sx_n = 0; sxx_n = 0;  sy_n = 0;	sxy_n = 0; syy_n = 0;

	for (j = 0; j < D; j++) {
                genekj[j]=gene[k][j];
		if(sample[j]=='1')
			sx +=  genekj[j];
		else
		        sx_n +=  genekj[j];
	}
	sx /= wid;

	sx_n/=(D-wid);

	for (j = 0; j < D; j++) {
		if(sample[j]=='1')
			sxx += (sx-genekj[j]) * (sx-genekj[j]);
		else
			sxx_n += (sx_n-genekj[j]) * (sx_n-genekj[j]);

	}

	sxx = (float)sqrt(sxx);
	sxx_n = (float)sqrt(sxx_n);

	for (j = 0; j < D; j++) {
                geneij[j]=gene[i][j];
		if(sample[j]=='1')
			sy +=  geneij[j];
		else
			sy_n +=  geneij[j];
	}

	sy /= wid; 

	sy_n /= (D-wid); 

	for (j = 0; j < D; j++)
	{

		if(sample[j]=='1') {

			sxy += (sx - genekj[j]) * (sy - geneij[j]);
			syy += (sy - geneij[j]) * (sy - geneij[j]);
		}
		else {

			sxy_n += (sx_n - genekj[j]) * (sy_n - geneij[j]);
			syy_n += (sy_n - geneij[j]) * (sy_n - geneij[j]);
		}

	}

	syy = (float)sqrt(syy);

	syy_n = (float)sqrt(syy_n);

	rval.r =  fabsf(sxy/(sxx * syy));

	rval.n_r =  fabsf(sxy_n/(sxx_n * syy_n));
        
        return rval;

}



__global__ void computebicluster_cu(float **gene, int n,int maxbcn,int D, float thr, char **maxbc_sample,char **maxbc_data,float *maxbc_score,int *maxbc_datacount, int *maxbc_samplecount, char **tmpbc_sample,char **tmpbc_data)
{

   int k=blockIdx.x*blockDim.x+threadIdx.x;

   if(k<maxbcn) {

                //extern __shared__ char arr[];

		float jcc,mean_k,mean_i;
		int i,j,l,vl,wid,wid_0,wid_1,wid_2,l_i,t_tot,t_dif;
		int dif,tot;
		//char *vect[3];
		__shared__  char vect[3][MAXSAMPLE];
		struct pair_r rval;
                int tmpbc_datacount,tmpbc_samplecount;

                float genekj,geneij;

                //for(i=0;i<3;i++)
                    //vect[i]=&arr[i*D];

		maxbc_score[k]=1.0;
		maxbc_datacount[k]=0;  

 
		//calculate mean expression for gene k

		mean_k=gene[k][D];

		for (i = k+1; i < n; i++) //pair k,i
		{ 	
			//calculate mean expression for gene i
			mean_i=gene[i][D];

			wid_0=0; wid_1=0; wid_2=0;      

			for (j = 0; j < D; j++)  
			{
                                genekj=gene[k][j];
                                geneij=gene[i][j];

				if ((genekj - mean_k)>=0 && (geneij - mean_i)>=0) //i and k upregulated : positive correlation
				{

					vect[0][j] = '1';
					vect[1][j] = '0';
					vect[2][j] = '0';

					wid_0+=1;
				}
				else if ((genekj - mean_k)<0 && (geneij - mean_i)<0)  // i and k down regulated : positive correlation
				{

					vect[0][j] = '0';
					vect[1][j] = '1';
					vect[2][j] = '0';

					wid_1+=1;
				}
				else if ((genekj - mean_k)*(geneij - mean_i)<0) //betwenn i and k one is up regulated and the other one is down regulated : negative correlation
				{

					vect[0][j] = '0';
					vect[1][j] = '0';
					vect[2][j] = '1';
					wid_2+=1;

				} 

			}

			for (vl = 0; vl < 3; vl++)
			{ 
				dif=0; tot=0;
                                if(vl==0)
                                    wid=wid_0; 
                                else if(vl==1)
                                    wid=wid_1; 
                                if(vl==2)
                                    wid=wid_2; 

				  
				if(wid>min) { //minimum samples required to form a bicluster module. Default minimum set to 10 in ccs.h   

					    rval=comput_r_cuda(vect[vl],wid, k, i, D, gene);
				}
				else {

					continue;
				}

				if (rval.r > thr) 
				{
					tot++;      
					if(rval.n_r>thr)
					    dif++;

					for (j = 0;j < D; j++)
						tmpbc_sample[k][j] = vect[vl][j];

					for (j = 0;j < n; j++)
						tmpbc_data[k][j] = '0';

					tmpbc_data[k][k] = '1';
					tmpbc_data[k][i] = '1';
					tmpbc_datacount = 2;
					tmpbc_samplecount = wid;


					for (l = 0; l < n; l++)  { //bicluster augmentation
						if (l != i && l != k) {
		                                        t_tot=0; t_dif=0;
		                                        for(l_i=0;l_i<n;l_i++) {
		                                                    if(tmpbc_data[k][l_i]=='1')  {
		                                                            rval=comput_r_cuda(vect[vl],wid, l, l_i, D, gene);
		                                                    
								            if(rval.r>thr) 
									               t_tot+=1;
		                                                            else {
		                                                                       t_tot=0;
		                                                                       break;
		                                                            }   
		                                                            if(rval.n_r>thr) 
										       t_dif+=1;
		                                                       }  
		                                         }                                                                    


							 if(t_tot>0)  {
		                                    	            tmpbc_data[k][l] = '1';
								    tmpbc_datacount+=1;
		                                                    tot+=t_tot; dif+=t_dif;
		                                          }
						}


					}  // end of augmentation

					// Compute Jaccard score

					if(tot>0)
					    jcc=(float)dif/tot;   
					else
					   jcc=1.0; 

					/*   Select bicluster candidate as the largest (maxbc[k].datacount<tmpbc.datacount) 
					     of all condition dependent (jaccard score <0.01) bicluster for k. Minimum number of gene 
		                             for a bicluster is set at 10. See the mingene at ccs.h                                */

					if(jcc<0.01 && maxbc_datacount[k]<tmpbc_datacount && tmpbc_datacount>mingene)
					{
						maxbc_score[k]=jcc;
						for (j = 0; j < n; j++)  
					       		maxbc_data[k][j]=tmpbc_data[k][j];
						for (j = 0; j < D; j++)  
					   		maxbc_sample[k][j]=tmpbc_sample[k][j];
						maxbc_datacount[k]=tmpbc_datacount;
						maxbc_samplecount[k]=tmpbc_samplecount;

					 }


				}    //end of r>thr condition
			}    //end of loop for vl	

		}  // end of i loop
         //for(i=0;i<3;i++)
              //free(vect[i]);

	}
}




int main(int argc, char *argv[])
{
	FILE   *in,*out;
	struct gn *gene;
	char **Hd;
	char *infile,*outfile;	
	int c, errflag,maxbcn=MAXB,print_type=0;
	int i,n,D;
	extern char *optarg;
	float thr,**device_gene,**device_gene_temp;
        struct bicl *bicluster;
        float overlap=100.0; 

        char **device_bicluster_sample,**device_bicluster_temp_sample,**device_bicluster_data,**device_bicluster_temp_data;
        float *device_bicluster_score,*device_bicluster_temp_score;

        int *device_bicluster_datacount,*device_bicluster_temp_datacount;
        int *device_bicluster_samplecount,*device_bicluster_temp_samplecount;

        char **device_bicluster_sample_tmpbc,**device_bicluster_temp_sample_tmpbc,**device_bicluster_data_tmpbc,**device_bicluster_temp_data_tmpbc;



	clock_t start = clock() ;
	in= out = NULL;

  	n = D = 0;
	thr = errflag = 0;
  
	while ((c = getopt(argc, argv, "ht:m:i:p:o:g:?")) != -1)
  	{
    		switch(c)
    		{
    			case 'h': // help
      				printUsage();
      				exit(0);
    			case 't': // threshold value
      				thr = atof(optarg);
      				break;
    			case 'm': // maximum number of bicluster search
      				maxbcn = atoi(optarg);
      				break;

    			case 'g': // output file format
      				overlap = atof(optarg);
     				break;

    			case 'p': // output file format
      				print_type = atoi(optarg);
     				break;

    			case 'i': // the input expression file
      				infile = optarg;
      				break;
    			case 'o': // the output file
      				outfile = optarg;
      				break;
    			case ':':       /* -f or -o without operand */
      				printf("Option -%c requires an operand\n", optopt);
      				errflag++;
     				break;
    			case '?':
      				fprintf(stderr,"Unrecognized option: -%c\n", optopt);
      				errflag++;
    		}
 	}



  	if (thr == 0)
  	{
    		fprintf(stderr,"***** WARNING: Threshold Theta (corr coeff) "
                   "value assumed to be ZERO (0)\n");
  	}


  	if (outfile[0] == '\0')
  	{
    		fprintf(stderr,"***** WARNING: Output file assumed to be STDOUT\n");
    		out = stdout;
  	}
  	else if ((out = fopen(outfile,"w")) == NULL) //write open bicluster file
  	{
    		fprintf(stderr,"***** ERROR: Unable to open Output file %s\n",outfile);
    		errflag++;
  	}

	if ((thr < 0) || (thr > 1))
  	{
    		fprintf(stderr,"***** ERROR: Threshold Theta (corr coeff) "
                   "must be between 0.0-1.0\n");
  	}

  	if (infile[0] == '\0')
  	{
    		fprintf(stderr,"***** ERROR: Input file not defined\n");
    		if (out) fclose(out);
    			errflag++;
  	}
 	else if ((in = fopen(infile,"r")) == NULL)  //open gene file
  	{
    		fprintf(stderr,"***** ERROR: Unable to open Input %s\n", infile);
    		if (out) fclose(out);
    			errflag++;
  	}

  	if (errflag)
  	{
    		printUsage();
    		exit(1);
  	}

	getmatrixsize(in,&n,&D);
	printf("Number of rows=%d\tNumber of columns=%d\n",n,D);

	if(maxbcn>n)
		maxbcn=n;

 	gene = (struct gn *)calloc(n,sizeof(struct gn));
 	Hd = (char **)calloc(D+1,sizeof(char *));

        hipMalloc((void **)&device_gene, n*sizeof(float *));
        device_gene_temp = (float **)calloc(n,sizeof(float *));

  	for (i = 0; i < n; i++) {
	    gene[i].x = ( float *)calloc(D+1,sizeof( float));
            hipMalloc( (void **)&device_gene_temp[i], (D+1)*sizeof(float));

	    if (!gene[i].x) {
		    printf("***** Memory ERROR: Can't allocate memory to read input data; Exiting \n");
		    exit(1);
   	    }
         }

  	readgene(infile,gene,Hd,n,D);	

  	for (i = 0; i < n; i++) {
              hipMemcpy(device_gene_temp[i],gene[i].x, (D+1)*sizeof(float), hipMemcpyHostToDevice);
        }  	

        hipMemcpy(device_gene, device_gene_temp, n*sizeof(float *), hipMemcpyHostToDevice);


	bicluster = (struct bicl *)calloc(maxbcn,sizeof(struct bicl));

        hipMalloc((void **)&device_bicluster_sample, maxbcn*sizeof(char *));
        device_bicluster_temp_sample = (char **)calloc(maxbcn,sizeof(char *));

        hipMalloc((void **)&device_bicluster_data, maxbcn*sizeof(char *));
        device_bicluster_temp_data = (char **)calloc(maxbcn,sizeof(char *));

        hipMalloc((void **)&device_bicluster_score, maxbcn*sizeof(float));
        device_bicluster_temp_score = (float *)calloc(maxbcn,sizeof(float));


        hipMalloc((void **)&device_bicluster_datacount, maxbcn*sizeof(int));
        device_bicluster_temp_datacount = (int *)calloc(maxbcn,sizeof(int));

        hipMalloc((void **)&device_bicluster_samplecount, maxbcn*sizeof(int));
        device_bicluster_temp_samplecount = (int *)calloc(maxbcn,sizeof(int));




        hipMalloc((void **)&device_bicluster_sample_tmpbc, maxbcn*sizeof(char *));
        device_bicluster_temp_sample_tmpbc = (char **)calloc(maxbcn,sizeof(char *));

        hipMalloc((void **)&device_bicluster_data_tmpbc, maxbcn*sizeof(char *));
        device_bicluster_temp_data_tmpbc = (char **)calloc(maxbcn,sizeof(char *));



        if (!bicluster) {
		    printf("***** Memory ERROR: Can't allocate memory for biclusters; Exiting \n");
		    exit(1);
	}



  	for (i = 0; i < maxbcn; i++)
  	{
		bicluster[i].sample = (char *)calloc(D,sizeof(char));
		bicluster[i].data = (char *)calloc(n,sizeof(char));

		hipMalloc( (void **)&device_bicluster_temp_sample[i], D*sizeof(char));
		hipMalloc( (void **)&device_bicluster_temp_data[i], n*sizeof(char));


		hipMalloc( (void **)&device_bicluster_temp_sample_tmpbc[i], D*sizeof(char));
		hipMalloc( (void **)&device_bicluster_temp_data_tmpbc[i], n*sizeof(char));

		if (!bicluster[i].sample || !bicluster[i].data) {
			    printf("***** Memory ERROR: Can't allocate memory for %d thbiclusters; Exiting \n",i+1);
			    exit(1);
		}

  	}
        
         
        hipMemcpy(device_bicluster_sample, device_bicluster_temp_sample, maxbcn*sizeof(char *), hipMemcpyHostToDevice);

        hipMemcpy(device_bicluster_data, device_bicluster_temp_data, maxbcn*sizeof(char *), hipMemcpyHostToDevice);


        hipMemcpy(device_bicluster_sample_tmpbc, device_bicluster_temp_sample_tmpbc, maxbcn*sizeof(char *), hipMemcpyHostToDevice);

        hipMemcpy(device_bicluster_data_tmpbc, device_bicluster_temp_data_tmpbc, maxbcn*sizeof(char *), hipMemcpyHostToDevice);

        //int sherememsize=(3*D)*(sizeof(char)); 

        //computebicluster_cu<<<maxbcn,1,sherememsize>>>(device_gene,n,maxbcn,D,thr,device_bicluster_sample,device_bicluster_data,device_bicluster_score,device_bicluster_datacount,device_bicluster_samplecount, device_bicluster_sample_tmpbc,device_bicluster_data_tmpbc);


        computebicluster_cu<<<maxbcn,1>>>(device_gene,n,maxbcn,D,thr,device_bicluster_sample,device_bicluster_data,device_bicluster_score,device_bicluster_datacount,device_bicluster_samplecount, device_bicluster_sample_tmpbc,device_bicluster_data_tmpbc);

        hipMemcpy(device_bicluster_temp_score, device_bicluster_score, maxbcn*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(device_bicluster_temp_datacount, device_bicluster_datacount, maxbcn*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(device_bicluster_temp_samplecount, device_bicluster_samplecount, maxbcn*sizeof(int), hipMemcpyDeviceToHost);


        for(i=0; i<maxbcn; i++)
        {
             hipMemcpy(bicluster[i].sample, device_bicluster_temp_sample[i], D*sizeof(char), hipMemcpyDeviceToHost);
             hipMemcpy(bicluster[i].data, device_bicluster_temp_data[i], n*sizeof(char), hipMemcpyDeviceToHost);

             bicluster[i].score=device_bicluster_temp_score[i];

             bicluster[i].datacount=device_bicluster_temp_datacount[i];
             bicluster[i].samplecount=device_bicluster_temp_samplecount[i];

         }



        printbicluster(out,gene,Hd,n,D,maxbcn,thr,bicluster,print_type,overlap);




  	for (i = 0; i < n; i++) {
                     
		free(gene[i].x);
		hipFree(device_gene_temp[i]);
        }

	for (i = 0; i < D+1; i++)   {   
		free(Hd[i]);
	}

	free(Hd);
	free(gene);
        free(device_gene_temp);
        hipFree(device_gene);

   
	for (i = 0; i < maxbcn; i++)
  	{  
		free(bicluster[i].sample);
		free(bicluster[i].data);
		hipFree(device_bicluster_temp_sample[i]);
		hipFree(device_bicluster_temp_data[i]);

		hipFree(device_bicluster_temp_sample_tmpbc[i]);
		hipFree(device_bicluster_temp_data_tmpbc[i]);

        }


	hipFree(device_bicluster_sample);
        hipFree(device_bicluster_data);
        hipFree(device_bicluster_score);
        hipFree(device_bicluster_datacount);
        hipFree(device_bicluster_samplecount);

	hipFree(device_bicluster_sample_tmpbc);
        hipFree(device_bicluster_data_tmpbc);



	free(device_bicluster_temp_data);
	free(device_bicluster_temp_sample);
	free(device_bicluster_temp_score);

	free(device_bicluster_temp_data_tmpbc);
	free(device_bicluster_temp_sample_tmpbc);

        free(bicluster);



 	clock_t end = clock() ;
 	float elapsed_time = (end-start)/( float)CLOCKS_PER_SEC ;
 	printf("Ellapsed time= %f\n",elapsed_time);
        if(print_type==0)     
        	fprintf(out,"\n\nEllapsed time= %f\n",elapsed_time);

  	if (out) fclose(out);

   	return SUCCESS;
}

void printUsage()
{
printf("\n\t\tUsage: ./ccs_cuda\n"
         "\t\t         -h [display this help message]\n"
         "\t\t         -t threshold theta in a range 0.0 - 1.0\n"
         "\t\t         -o output file\n"
         "\t\t         -i input microarray expression file (processed data)\n"
         "\t\t         -m optional parameter maximum expected biclusters in a range 1 - number_of_rows_in_input_data_matrix with a default 1000\n"
         "\t\t         -g optional parameter minimum percentage of gene overlap between merged biclusters in a range 0-100 with a default 100 percent for full overlap\n"
         "\t\t         -p optional output file format : 0 for standard format, 1 for BiBench bicluster format and default is 0\n"
        );
}

